#include "hip/hip_runtime.h"
/**
* @file hermes.cu
* @brief Hermes emulation
* @author John Melton, G0ORX/N6LYT
*/


/* Copyright (C)
* 2015 - John Melton, G0ORX/N6LYT
*
* Based on code by Steven Passe AD0ES and Vasiliy Gokoyev K3IT
*
* This program is free software; you can redistribute it and/or
* modify it under the terms of the GNU General Public License
* as published by the Free Software Foundation; either version 2
* of the License, or (at your option) any later version.
*
* This program is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
* GNU General Public License for more details.
*
* You should have received a copy of the GNU General Public License
* along with this program; if not, write to the Free Software
* Foundation, Inc., 59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.
*
*/

#include <errno.h>
#include <pthread.h>
#include <sched.h>
#include <semaphore.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <sys/ioctl.h>
#include <sys/socket.h>
#include <arpa/inet.h>
#include <net/if.h>
#include <netinet/if_ether.h>
#include <netpacket/packet.h>
#include <net/if_packet.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime_api.h>

#include "common.cuh"
#include "dfc.cuh"
#include "inputbuffer.cuh"
#include "rawbuffer.cuh"
#include "filters.cuh"
#include "receiver.cuh"
#include "hermes.cuh"
#include "time.cuh"
#include "audio.cuh"

#define PORT 1024
#define MAX_BUFFER_LEN 1032

#define HERMES_FW_VERSION 26
#define HERMES_ID 0x01
#define HERMES_LITE_ID 0x06

#define SYN 0x7f



//#define SCALE_FACTOR  0x7fffffffL
#define SCALE_FACTOR  8388607.0 // 2^24-1

int hpsdr_id=HERMES_ID;

static int slicesamples;

static pthread_t readThreadId;
static pthread_t processThreadId;
static pthread_t processRawThreadId;

static int hermesSocket;

static unsigned char hw_address[6];

static int state=0; // 0 = idle, 1 = running
struct sockaddr_in clientAddr;

static int sendIQ=0;
static int sendRaw=0;

static int outputrate=-1; // nothing
static int outputsamplerate=0;
static int receivers=1;
static int mox=0;
static int commonfrequency=0;

static long tx_sequence=0;
static long raw_sequence=0;

#define MAX_RECEIVERS 7
/*
static long frequency[MAX_RECEIVERS] = {14150000,14150000,14150000,14150000,14150000,14150000,14150000};
static int rotate[MAX_RECEIVERS] = {0,0,0,0,0,0,0};
static hipfftComplex* receiverdata[MAX_RECEIVERS];
static hipfftComplex* deviceReceiverdata[MAX_RECEIVERS];
static hipfftComplex* slicedata[MAX_RECEIVERS];
static hipfftComplex* deviceSlicedata[MAX_RECEIVERS];
static hipfftComplex* slice[MAX_RECEIVERS];
static hipfftComplex* deviceSlice[MAX_RECEIVERS];
static hipfftComplex* decimate[MAX_RECEIVERS];
static hipfftComplex* deviceDecimate[MAX_RECEIVERS];
*/

static hipfftComplex* filter;
static hipfftComplex* deviceFilter;

static char* output[MAX_RECEIVERS];
static char* deviceOutput[MAX_RECEIVERS];

static float scale;
static int d_size;
static int d_size_2;
static int ifft_decimate_factor;
static int outrot;

static hipfftHandle planC2C;

#define FRAME_LENGTH 1032
static unsigned char frame[FRAME_LENGTH];
static int frameoffset;
static unsigned char rawframe[FRAME_LENGTH];
static int rawframeoffset;

void* readThread(void* arg);
void* processThread(void* arg);
void* processRawThread(void* arg);
void processClientData(unsigned char* buffer);
void processClientFrame(unsigned char* buffer);

void initHermes() {
    int result;
    hipError_t error;

    fprintf(stderr,"initHermes\n");

    scale=1.0;

    for(int i=0;i<FRAME_LENGTH;i++) {
        frame[i]='\0';
    }

    frame[0]=0xef;
    frame[1]=0xfe;
    frame[2]=0x01;
    frame[3]=0x06;
    frame[4]=0x00;
    frame[5]=0x00;
    frame[6]=0x00;
    frame[7]=0x00;

    frame[8]=0x7f;
    frame[9]=0x7f;
    frame[10]=0x7f;
    frame[11]=0x00;
    frame[12]=0x1e;
    frame[13]=0x00;
    frame[14]=0x00;
    frame[15]=HERMES_FW_VERSION;

    frame[520]=0x7f;
    frame[521]=0x7f;
    frame[522]=0x7f;
    frame[523]=0x00;
    frame[524]=0x1e;
    frame[525]=0x00;
    frame[526]=0x00;
    frame[527]=HERMES_FW_VERSION;

    frameoffset=16;

    rawframe[0]=0xef;
    rawframe[1]=0xfe;
    rawframe[2]=0x01;
    rawframe[3]=0x04;
    rawframe[4]=0x00;
    rawframe[5]=0x00;
    rawframe[6]=0x00;
    rawframe[7]=0x00;
    rawframeoffset=8;

    if((result=pthread_create(&readThreadId, NULL, readThread, NULL)) < 0) {
        fprintf(stderr, "readThread create failed %d\n",result);
        exit(EXIT_FAILURE);
    }

    if((result=pthread_create(&processThreadId, NULL, processThread, NULL)) < 0) {
        fprintf(stderr, "processThread create failed %d\n",result);
        exit(EXIT_FAILURE);
    }

    if((result=pthread_create(&processRawThreadId, NULL, processRawThread, NULL)) < 0) {
        fprintf(stderr, "processRawThread create failed %d\n",result);
        exit(EXIT_FAILURE);
    }

}

void* readThread(void* arg) {
    struct sockaddr_in readAddr;
    uint8_t readBuffer[MAX_BUFFER_LEN];
    socklen_t readLength;
    struct ifreq ifr;

    readLength = sizeof(readAddr);

    fprintf(stderr,"hermes readThread: running on cpu %d\n", sched_getcpu());

    hermesSocket = socket(PF_INET, SOCK_DGRAM, IPPROTO_UDP);
    if (hermesSocket < 0) {
        perror("readThread: create socket failed for hermesSocket\n");
        exit(EXIT_FAILURE);
    }

    int on=1;
    int rc = setsockopt(hermesSocket, SOL_SOCKET, SO_REUSEADDR, (const void*)&on, sizeof(on));
    if (rc != 0) {
        fprintf(stderr, "readThread: cannot set SO_REUSEADDR: rc=%d\n", rc);
        exit(EXIT_FAILURE);
    }

    // Bind to this interface.
    readAddr.sin_family = AF_INET;
    readAddr.sin_port = htons(PORT);
    readAddr.sin_addr.s_addr = htonl(INADDR_ANY);

    if (bind(hermesSocket, (struct sockaddr*) &readAddr, sizeof(readAddr)) < 0) {
        perror("readThread: bind socket failed for hermesSocket\n");
        exit(EXIT_FAILURE);
    }

    // Allow broadcast on the socket.
    rc = setsockopt(hermesSocket, SOL_SOCKET, SO_BROADCAST, (const void*)&on, sizeof(on));
    if (rc != 0) {
        fprintf(stderr, "readThread: cannot set SO_BROADCAST: rc=%d\n", rc);
        exit(EXIT_FAILURE);
    }

    ifr.ifr_addr.sa_family = AF_INET;
    strncpy(ifr.ifr_name, interface, IFNAMSIZ-1);
    ioctl(hermesSocket, SIOCGIFADDR, &ifr);

    unsigned char* u = (unsigned char*)&ifr.ifr_addr.sa_data;
    for (int k = 0; k < 6; k++) hw_address[k] = u[k];

    fprintf(stderr, "readThread: listening on %s (%02x:%02x:%02x:%02x:%02x:%02x)\n",
           inet_ntoa(((struct sockaddr_in *)&ifr.ifr_addr)->sin_addr),
           hw_address[0], hw_address[1], hw_address[2],
           hw_address[3], hw_address[4], hw_address[5]);

    unsigned char discoverBuffer[MAX_BUFFER_LEN] =
           { 0xef, 0xfe, 0x02, 0, 0, 0, 0, 0, 0, HERMES_FW_VERSION, hpsdr_id };

    while(1) {

        if ((rc=recvfrom(hermesSocket, readBuffer, sizeof(readBuffer), 0,
                      (struct sockaddr*)&readAddr, &readLength)) < 0) {
            fprintf(stderr, "readThread: Bad recvfrom %d", rc);
            exit(EXIT_FAILURE);
        }

        //fprintf(stderr,"recvfrom: %d bytes\n", rc);

        if ((readBuffer[0] == 0xef) && (readBuffer[1] == 0xfe))  {
            switch(readBuffer[2]) {
                case 1:
                    // data
                    if(state) {
                        // check if from expected client
                        if(memcmp(&clientAddr,&readAddr, readLength)==0) {
                            processClientData(readBuffer);
                        } else {
                            // ignore
                        }
                    } else {
                        processClientData(readBuffer);
                    }
                    break;
                case 2:
                    fprintf(stderr, "readThread: received discovery from %s %d\n",
                        inet_ntoa(readAddr.sin_addr), htons(readAddr.sin_port));
                    for (int i = 0; i < 6; i++) {
                        discoverBuffer[3 + i] = hw_address[i];
                    }
                    discoverBuffer[2] |= state;
                    discoverBuffer[10]=hpsdr_id;
                    for (int i = 11; i < 60; i++)
                        discoverBuffer[i] = 0;
                    if ((rc=sendto(hermesSocket, discoverBuffer, 60, 0,
                               (struct sockaddr*)&readAddr, sizeof(readAddr))) < 0) {
                        fprintf(stderr, "readThread: Bad sendto %d",rc);
                        exit(EXIT_FAILURE);
                    }
                   break;
               case 4:
                   // start/stop command
                   switch(readBuffer[3]) {
                       case 0:
                           if(state==0)  {
                               fprintf(stderr,"readThread: ignoring stop command from %s\n",
                                   inet_ntoa(readAddr.sin_addr));
                           } else if(memcmp(&clientAddr,&readAddr, readLength)==0) {
                               state=0;
                               sendIQ=0;
                               sendRaw=0;
                               tx_sequence=0;
                               raw_sequence=0;
                           } else {
                               fprintf(stderr,"readThread: ignoring stop command from %s\n",
                                   inet_ntoa(readAddr.sin_addr));
                           }
                           break;
                       case 1:
                           if(state==0)  {
                               memcpy(&clientAddr,&readAddr, readLength);
                               state=1;
                               sendIQ=1;
                               sendRaw=0;
                           } else if(memcmp(&clientAddr,&readAddr, readLength)==0) {
                               sendIQ=1;
                               sendRaw=0;
                           } else {
                               fprintf(stderr,"readThread: ignoring start command %d from %s\n",
                                   readBuffer[3], inet_ntoa(readAddr.sin_addr));
                           }
                           break;
                       case 2:
                           if(state==0)  {
                               memcpy(&clientAddr,&readAddr, readLength);
                               state=1;
                               sendIQ=0;
                               sendRaw=1;
                           } else if(memcmp(&clientAddr,&readAddr, readLength)==0) {
                               sendIQ=0;
                               sendRaw=1;
                           } else {
                               fprintf(stderr,"readThread: ignoring start command %d from %s\n",
                                   readBuffer[3], inet_ntoa(readAddr.sin_addr));
                           }
                           break;
                       case 3:
                           if(state==0)  {
                               memcpy(&clientAddr,&readAddr, readLength);
                               state=1;
                               sendIQ=1;
                               sendRaw=1;
                           } else if(memcmp(&clientAddr,&readAddr, readLength)==0) {
                               sendIQ=1;
                               sendRaw=1;
                           } else {
                               fprintf(stderr,"readThread: ignoring start command %d from %s\n",
                                   readBuffer[3], inet_ntoa(readAddr.sin_addr));
                           }
                           break;
                   }
                   fprintf(stderr,"readThread: received start/stop command: state=%d sendIQ=%d sendRaw=%d\n",
                           state, sendIQ, sendRaw);
                   break;
                default:
                   break;
            }
        } else {
            fprintf(stderr, "readThread: unexpected packet from %s (0x%02x 0x%02x 0x%02x)\n",
                inet_ntoa(readAddr.sin_addr),
                readBuffer[0], readBuffer[1], readBuffer[2]);
        }
    }
    
}

void processClientData(unsigned char* buffer) {
    int ep=buffer[3]&0xFF;
    if(ep==2) {
        processClientFrame(&buffer[8]);
        processClientFrame(&buffer[520]);
    } else {
        fprintf(stderr,"processClientData: unexpected endpoint %d\n", ep);
    }
}


void processClientFrame(unsigned char* buffer) {
    int id;
    int rate;
    int rcvrs;
    int rx;
    long f;
    int rot;
    hipError_t error;
    hipfftResult cufftError;

    if(buffer[0]==SYN && buffer[1]==SYN && buffer[2]==SYN) {
        mox=buffer[3]&0x01;
        id=(buffer[3]&0xFF)>>1;
        switch(id) {
            case 0:
                rate=buffer[4]&0x03;
                if(rate!=outputrate) {
                    outputrate=rate;
                    switch(rate) {
                        case 0:
                            outputsamplerate=48000;
                            break;
                        case 1:
                            outputsamplerate=96000;
                            break;
                        case 2:
                            outputsamplerate=192000;
                            break;
                        case 3:
                            outputsamplerate=384000;
                            break;
                    }
                    fprintf(stderr,"outputsamplerate=%d\n",outputsamplerate);

                    filter=getFilter(rate);
                    deviceFilter=getDeviceFilter(rate);

                    slicesamples=(int)((float)outputsamplerate/hzperbin);


                    //if(source=SOURCE_PCIE) {
                    //    d_size=256;
                    //    d_size_2=(samplingrate/256)/outputsamplerate;
                    //} else {
                        d_size=(samplingrate/10)/outputsamplerate;
                    //}

                    fprintf(stderr,"d_size=%d\n",d_size);

                    ifft_decimate_factor = (samplingrate / d_size / 2 / outputsamplerate);
                    fprintf(stderr,"ifft_decimate_factor=%d\n",ifft_decimate_factor);

                    outrot = (int)(round((outputsamplerate/2) * NFACTOR) * V_SIZE)+9;
                    fprintf(stderr,"outrot=%d\n",outrot);

                      
                    fprintf(stderr,"planC2C=%d\n",COMPLEX_SIGNAL_SIZE/d_size);
                    cufftError = hipfftPlan1d(&planC2C, COMPLEX_SIGNAL_SIZE/d_size, HIPFFT_C2C, 1);
                    if(cufftError!=HIPFFT_SUCCESS) {
                        fprintf(stderr,"processClientFrame: Error creating hipfftPlan1d for Inverse FFT: %s\n", _cudaGetErrorEnum(cufftError));
                        exit(EXIT_FAILURE);
                    }


fprintf(stderr,"P_SIZE:%d V_SIZE:%d L_SIZE:%d RX_TD_SIZE=%d\n", P_SIZE,V_SIZE,L_SIZE,RX_TD_SIZE);
fprintf(stderr,"DFT_BLOCK_SIZE:%d COMPLEX_SIGNAL_SIZE:%d\n", DFT_BLOCK_SIZE,COMPLEX_SIGNAL_SIZE);

                    for(int i=0;i<MAX_RECEIVER;i++) {
                        RECEIVER* r=&receiver[i];
                        r->outputrate=outputsamplerate;
                        r->filter=filter;
                        r->deviceFilter=deviceFilter;
                        r->slicesamples=slicesamples;
                        r->d_size=d_size;
                        r->ifft_decimate_factor=ifft_decimate_factor;
                        r->rx_td_size=RX_TD_SIZE;
                        r->planC2C=planC2C;
                        r->scale=1.0F;
                        r->outrot=outrot;
                        initReceiver(i);
                     }

                }
                rcvrs=((buffer[7]>>3)&0x07)+1;
                if(receivers!=rcvrs) {
                    receivers=rcvrs;
                    fprintf(stderr,"processClientFrame: setting receivers to %d\n", receivers);
                }

                commonfrequency=(buffer[7]>>7)&0x01;
                break;
            case 1: // tx frequency
                break;
            case 2:
            case 3:
            case 4:
            case 5:
            case 6:
            case 7:
            case 8:
                rx=id-2;
                f = (buffer[4]&0xFF) << 24 | (buffer[5]&0xFF) << 16 | (buffer[6]&0xFF) << 8 | (buffer[7]&0xFF);
                rot=(int)((((float)f-((float)outputsamplerate/2.0f))/hzperbin)+0.5f);
                //rot=(int)((((float)f-((float)outputsamplerate/2.0f))/hzperbin));
                //rot=(int)(((float)f/hzperbin)+0.5f);
                if(commonfrequency) {
                    for(rx=0;rx<receivers;rx++) {
                        receiver[rx].frequency=f;
                        receiver[rx].rotate=rot;
                    }
                } else {
                    receiver[rx].frequency=f;
                    receiver[rx].rotate=rot;
                }

                //fprintf(stderr,"set new frequency(%d) %ld rot=%d\n", rx, f, rot);
                break;
            case 9:
            case 10:
            case 11:
            case 12:
            case 13:
            case 14:
            case 15:
            case 16:
                break;
            default:
                break;
        }

        if(audio) {
            audio_write(buffer);
        }
    } else {
        fprintf(stderr,"processClientFrame: syn error 0x%02x 0x%02x 0x%02x\n", 
                buffer[0], buffer[1],buffer[2]);
    }
}

void* processThread(void* arg) {
    int result;
    hipError_t error;

#ifdef TIMING
    long long starttime;
    long long endtime;
#endif

    fprintf(stderr,"hermes processThread: running on cpu %d\n", sched_getcpu());

    // get the next buffer
    result=sem_post(&frequencyBufferEmpty);
    if(result!=0) {
        fprintf(stderr, "processThread: sem_post failed for frequencyBufferEmpty: %d\n", result);
        exit(EXIT_FAILURE);
    }

    while(1) {

        result=sem_wait(&frequencyBufferFull);
        if(result!=0) {
            fprintf(stderr, "processThread: sem_wait failed for frequencyBufferFull: %d\n", result);
            exit(EXIT_FAILURE);
        }

        if(state && sendIQ) {
#ifdef TIMING
            starttime=current_timestamp();
#endif
            // process the buffer for each receiver
            // TODO handle commonfrequency
            for(int i=0;i<receivers;i++) {
                result=sem_post(&receiver[i].inputReady);
                if(result!=0) {
                    fprintf(stderr, "processRawThread: sem_post failed for inputReady %d: %d\n", i, result);
                    exit(EXIT_FAILURE);
                }
            }

            for(int i=0;i<receivers;i++) {
                result=sem_wait(&receiver[i].outputReady);
                if(result!=0) {
                    fprintf(stderr, "processRawThread: sem_wait failed for inputReady %d: %d\n", i, result);
                    exit(EXIT_FAILURE);
                }
            }
  
            // can get the next buffer
            result=sem_post(&frequencyBufferEmpty);
            if(result!=0) {
                fprintf(stderr, "processThread: sem_post failed for frequencyBufferEmpty: %d\n", result);
                exit(EXIT_FAILURE);
            }

            // copy the IQ samples
//fprintf(stderr,"copying %d IQ samples\n", RX_TD_SIZE);
            for(int i=0;i<RX_TD_SIZE;i++) {
                // I/Q samples for each receiver
                for(int r=0;r<receivers;r++) {
                    for(int j=0;j<6;j++) {
                        frame[frameoffset++]=receiver[r].output[(i*6)+j];
                    }
                }
                // mic samples
                frame[frameoffset++]=0x00;
                frame[frameoffset++]=0x00;

                if(frameoffset<=520) {
                    if(frameoffset+(receivers*6)+2>520) {
//fprintf(stderr,"frameoffset=%d setting to 528\n",frameoffset);
                        frameoffset=528;
                    }
                } else if(frameoffset<=1032) {
                    if(frameoffset+(receivers*6)+2>1032) {
//fprintf(stderr,"frameoffset=%d sendign and setting to 16\n",frameoffset);
                        // send the frame
                        frame[4] = (tx_sequence >> 24) & 0xff;
                        frame[5] = (tx_sequence >> 16) & 0xff;
                        frame[6] = (tx_sequence >> 8) & 0xff;
                        frame[7] = tx_sequence & 0xff;


//fprintf(stderr,"send frame offset=%d seq=%ld\n",frameoffset,tx_sequence);
                        if ((result=sendto(hermesSocket, frame, 1032, 0,
                               (struct sockaddr*)&clientAddr, sizeof(clientAddr))) < 0) {
                            fprintf(stderr, "Error sending data to client %d\n", result);
                            exit(EXIT_FAILURE);
                        }

                        tx_sequence++;
                        frameoffset=16;
                    }
                }
            }

//fprintf(stderr,"copied samples: frameoffset=%d\n",frameoffset);
#ifdef TIMING
            endtime=current_timestamp();
            fprintf(stderr,"process took %lld ms to process %d receivers\n", endtime-starttime, receivers);
#endif

        } else {
            // can get the next buffer
            result=sem_post(&frequencyBufferEmpty);
            if(result!=0) {
                fprintf(stderr, "processThread: sem_post failed for frequencyBufferEmpty: %d\n", result);
                exit(EXIT_FAILURE);
            }

        }

    }
}

void* processRawThread(void* arg) {
    int result;
    fprintf(stderr,"hermes processRawThread: running on cpu %d\n", sched_getcpu());
    while(1) {
        // get the next buffer
        result=sem_post(&rawBufferEmpty);
        if(result!=0) {
            fprintf(stderr, "processRawThread: sem_post failed for rawBufferEmpty: %d\n", result);
            exit(EXIT_FAILURE);
        }

        result=sem_wait(&rawBufferFull);
        if(result!=0) {
            fprintf(stderr, "processRawThread: sem_wait failed for rawBufferFull: %d\n", result);
            exit(EXIT_FAILURE);
        }

        if(state && sendRaw) {
            for(int i=0;i<RAW_BUFFER_SIZE;i++) {
                rawframe[rawframeoffset++]=rawBuffer[i]&0xFF;
                rawframe[rawframeoffset++]=(rawBuffer[i]>>8)&0xFF;
                if(rawframeoffset>=1032) {

                    rawframe[4] = (raw_sequence >> 24) & 0xff;
                    rawframe[5] = (raw_sequence >> 16) & 0xff;
                    rawframe[6] = (raw_sequence >> 8) & 0xff;
                    rawframe[7] = raw_sequence & 0xff;

                    if ((result=sendto(hermesSocket, rawframe, 1032, 0,
                           (struct sockaddr*)&clientAddr, sizeof(clientAddr))) < 0) {
                        fprintf(stderr, "Error sending raw data to client %d\n", result);
                        exit(EXIT_FAILURE);
                    }

                    raw_sequence++;
                    rawframeoffset=8;
                }
            }
        }
        
    }
}
