/**
* @file time.cu
* @brief Timestamping
* @author John Melton, G0ORX/N6LYT
*/


/* Copyright (C)
* 2015 - John Melton, G0ORX/N6LYT
*
* Based on code by Steven Passe AD0ES and Vasiliy Gokoyev K3IT
*
* This program is free software; you can redistribute it and/or
* modify it under the terms of the GNU General Public License
* as published by the Free Software Foundation; either version 2
* of the License, or (at your option) any later version.
*
* This program is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
* GNU General Public License for more details.
*
* You should have received a copy of the GNU General Public License
* along with this program; if not, write to the Free Software
* Foundation, Inc., 59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.
*
*/


#include <hip/hip_runtime.h>
#include <sys/time.h>

long long current_timestamp() {
    struct timeval te; 
    gettimeofday(&te, NULL);
    long long milliseconds = te.tv_sec*1000LL + te.tv_usec/1000;
    return milliseconds;
}
