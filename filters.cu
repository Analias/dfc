#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime_api.h>

#include "common.cuh"
#include "filters.cuh"
#include "coeff.cuh"

hipfftComplex* filter48k;
hipfftComplex* filter96k;
hipfftComplex* filter192k;
hipfftComplex* filter384k;

hipfftComplex* deviceFilter48k;
hipfftComplex* deviceFilter96k;
hipfftComplex* deviceFilter192k;
hipfftComplex* deviceFilter384k;

//static hipfftHandle      planR2C;

__global__ void	gpu_make_analytic(hipfftComplex*);

void generateFilter(double samplingrate,double Fb,hipfftComplex* deviceFilter,hipfftComplex* filter);

void loadFilters(int pcie) {
    
fprintf(stderr,"filter size=%ld\n",(COMPLEX_SIGNAL_SIZE+1)*sizeof(hipfftComplex));

    hipError_t result = hipHostAlloc(&filter48k, (COMPLEX_SIGNAL_SIZE+1)*sizeof(hipfftComplex), hipHostMallocMapped);
    if (result != hipSuccess) {
       fprintf(stderr, "Error filter 48k hipHostAlloc %d\n", result);
       exit(EXIT_FAILURE);
    }
    result = hipHostGetDevicePointer(&deviceFilter48k, filter48k, 0);
    if (result != hipSuccess) {
       fprintf(stderr, "Error filter 48k hipHostGetDevicePointer %d\n", result);
       exit(EXIT_FAILURE);
    }
    generateFilter(samplingrate,48000.0,deviceFilter48k,filter48k);

    result = hipHostAlloc(&filter96k, (COMPLEX_SIGNAL_SIZE+1)*sizeof(hipfftComplex), hipHostMallocMapped);
    if (result != hipSuccess) {
       fprintf(stderr, "Error filter 96k hipHostAlloc %d\n", result);
       exit(EXIT_FAILURE);
    }
    result = hipHostGetDevicePointer(&deviceFilter96k, filter96k, 0);
    if (result != hipSuccess) {
       fprintf(stderr, "Error filter 96k hipHostGetDevicePointer %d\n", result);
       exit(EXIT_FAILURE);
    }
    generateFilter(samplingrate,96000.0,deviceFilter96k,filter96k);

    result = hipHostAlloc(&filter192k, (COMPLEX_SIGNAL_SIZE+1)*sizeof(hipfftComplex), hipHostMallocMapped);
    if (result != hipSuccess) {
       fprintf(stderr, "Error filter 192k hipHostAlloc %d\n", result);
       exit(EXIT_FAILURE);
    }
    result = hipHostGetDevicePointer(&deviceFilter192k, filter192k, 0);
    if (result != hipSuccess) {
       fprintf(stderr, "Error filter 192k hipHostGetDevicePointer %d\n", result);
       exit(EXIT_FAILURE);
    }
    generateFilter(samplingrate,192000.0,deviceFilter192k,filter192k);

    result = hipHostAlloc(&filter384k, (COMPLEX_SIGNAL_SIZE+1)*sizeof(hipfftComplex), hipHostMallocMapped);
    if (result != hipSuccess) {
       fprintf(stderr, "Error filter 384k hipHostAlloc %d\n", result);
       exit(EXIT_FAILURE);
    }
    result = hipHostGetDevicePointer(&deviceFilter384k, filter384k, 0);
    if (result != hipSuccess) {
       fprintf(stderr, "Error filter 384k hipHostGetDevicePointer %d\n", result);
       exit(EXIT_FAILURE);
    }
    generateFilter(samplingrate,384000.0,deviceFilter384k,filter384k);
}

void generateFilter(double samplingrate,double Fb,hipfftComplex* deviceFilter,hipfftComplex* filter) {

    hipfftReal* coeff;
    hipfftReal* deviceCoeff;
    hipfftHandle planR2C;

fprintf(stderr,"coeffs size=%ld\n",DFT_BLOCK_SIZE*sizeof(hipfftReal));
    hipError_t result = hipHostAlloc(&coeff, DFT_BLOCK_SIZE*sizeof(hipfftReal), hipHostMallocMapped);
    if (result != hipSuccess) {
       fprintf(stderr, "Error coeff hipHostAlloc %d\n", result);
       exit(EXIT_FAILURE);
    }

    result = hipHostGetDevicePointer(&deviceCoeff, coeff, 0);
    if (result != hipSuccess) {
       fprintf(stderr, "Error coeff hipHostGetDevicePointer %d\n", result);
       exit(EXIT_FAILURE);
    }

    double *H=calcFilter(samplingrate,0.0,Fb,P_SIZE,25.0);
    for(int i=0;i<P_SIZE;i++) {
        coeff[i]=(hipfftReal)H[i];
    }
    free(H);

    for(int i=P_SIZE; i<DFT_BLOCK_SIZE;i++) {
        coeff[i]=0.0;
    }

#ifdef DUMPDATA
for(int i=0;i<1024;i++) {
    fprintf(stderr,"coeff %d=%.24f\n", i, coeff[i]);
}
#endif

    hipfftResult error = hipfftPlan1d(&planR2C, DFT_BLOCK_SIZE, HIPFFT_R2C, 1);
    if(error!=HIPFFT_SUCCESS) {
       fprintf(stderr,"Error creating hipfftPlan1d for FIR: %s\n", _cudaGetErrorEnum(error));
       exit(EXIT_FAILURE);
    }

    error = hipfftExecR2C(planR2C, deviceCoeff, deviceFilter);
    if (error != HIPFFT_SUCCESS) {
       fprintf(stderr, "Error hipfftExecR2C (planR2C) %d\n", result);
       exit(EXIT_FAILURE);
    }

    gpu_make_analytic<<<COMPLEX_SIGNAL_SIZE/4096, 1024>>>(deviceFilter);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if(err != hipSuccess)
    {
      // print the CUDA error message and exit
      fprintf(stderr,"gpu_make_analytic error: %s\n", hipGetErrorString(err));
      exit(-1);
    }


#ifdef DUMPDATA
for(int i=0;i<1024;i++) {
    fprintf(stderr,"analytic %d=%.24f:%.24f\n", i, filter[i].x, filter[i].y);
}
#endif

    error = hipfftDestroy(planR2C);
    if (error != HIPFFT_SUCCESS) {
       fprintf(stderr, "Error hipfftDestroy (planR2C) %d\n", result);
       exit(EXIT_FAILURE);
    }

    result = hipHostFree(coeff);
    if (result != hipSuccess) {
       fprintf(stderr, "Error coeff hipHostFree %d\n", result);
       exit(EXIT_FAILURE);
    }
}

hipfftComplex* getFilter(int rate) {
    hipfftComplex* filter;
    filter=(hipfftComplex*)0;
    switch(rate) {
        case 0:
           filter=filter48k;
           break;
        case 1:
           filter=filter96k;
           break;
        case 2:
           filter=filter192k;
           break;
        case 3:
           filter=filter384k;
           break;
        default:
           fprintf(stderr,"getFilter: invalid rate %d/n", rate);
           exit(EXIT_FAILURE);
           break;
    }
    return filter;
}

hipfftComplex* getDeviceFilter(int rate) {
    hipfftComplex* filter;
    filter=(hipfftComplex*)0;
    switch(rate) {
        case 0:
           filter=deviceFilter48k;
           break;
        case 1:
           filter=deviceFilter96k;
           break;
        case 2:
           filter=deviceFilter192k;
           break;
        case 3:
           filter=deviceFilter384k;
           break;
        default:
           fprintf(stderr,"getDeviceFilter: invalid rate %d/n", rate);
           exit(EXIT_FAILURE);
           break;
    }
    return filter;
}

