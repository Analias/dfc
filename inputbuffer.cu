#include <stdio.h>
#include <stdlib.h>
#include <pthread.h>
#include <semaphore.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime_api.h>

#include "common.cuh"
#include "inputbuffer.cuh"


short* inputBuffer;
short* deviceInputBuffer;

sem_t inputBufferEmpty;
sem_t inputBufferFull;

void initInputBuffer() {
    fprintf(stderr,"initInputBuffer: size=%ld\n",L_SIZE*sizeof(short));
    hipError_t result = hipHostAlloc(&inputBuffer, L_SIZE*sizeof(short), hipHostMallocMapped);
    if (result != hipSuccess) {
       fprintf(stderr, "Error inputBuffer hipHostAlloc %d\n", result);
       exit(EXIT_FAILURE);
    }

    result = hipHostGetDevicePointer(&deviceInputBuffer, inputBuffer, 0);
    if (result != hipSuccess) {
       fprintf(stderr, "Error inputBuffer hipHostGetDevicePointer %d\n", result);
       exit(EXIT_FAILURE);
    }


    int res=sem_init(&inputBufferEmpty, 0, 0);
    if(result!=0) {
        fprintf(stderr,"inputbuffer: sem_init failed for inputBufferEmpty%d\n", result);
        exit(EXIT_FAILURE);
    }

    res=sem_init(&inputBufferFull, 0, 0);
    if(result!=0) {
        fprintf(stderr,"inputbuffer: sem_init failed for inputBufferFull%d\n", result);
        exit(EXIT_FAILURE);
    }

}
