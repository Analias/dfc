#include <stdio.h>
#include <stdlib.h>
#include <pthread.h>
#include <semaphore.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime_api.h>

#include "rawbuffer.cuh"

short* rawBuffer;
short* deviceRawBuffer;

sem_t rawBufferEmpty;
sem_t rawBufferFull;

void initRawBuffer() {
    fprintf(stderr,"initRawBuffer\n");
    hipError_t result = hipHostAlloc(&rawBuffer, RAW_BUFFER_SIZE*sizeof(short), hipHostMallocMapped);
    if (result != hipSuccess) {
       fprintf(stderr, "Error rawBuffer hipHostAlloc %d\n", result);
       exit(EXIT_FAILURE);
    }

    result = hipHostGetDevicePointer(&deviceRawBuffer, rawBuffer, 0);
    if (result != hipSuccess) {
       fprintf(stderr, "Error rawBuffer hipHostGetDevicePointer %d\n", result);
       exit(EXIT_FAILURE);
    }


    int res=sem_init(&rawBufferEmpty, 0, 0);
    if(result!=0) {
        fprintf(stderr,"rawbuffer: sem_init failed for rawBufferEmpty%d\n", result);
        exit(EXIT_FAILURE);
    }

    res=sem_init(&rawBufferFull, 0, 0);
    if(result!=0) {
        fprintf(stderr,"rawbuffer: sem_init failed for rawBufferFull%d\n", result);
        exit(EXIT_FAILURE);
    }

}
