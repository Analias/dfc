#include "hip/hip_runtime.h"
/**
* @file filters.cu
* @brief Bandpass filters
* @author John Melton, G0ORX/N6LYT
*/


/* Copyright (C)
* 2015 - John Melton, G0ORX/N6LYT
*
* Based on code by Steven Passe AD0ES and Vasiliy Gokoyev K3IT
*
* This program is free software; you can redistribute it and/or
* modify it under the terms of the GNU General Public License
* as published by the Free Software Foundation; either version 2
* of the License, or (at your option) any later version.
*
* This program is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
* GNU General Public License for more details.
*
* You should have received a copy of the GNU General Public License
* along with this program; if not, write to the Free Software
* Foundation, Inc., 59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.
*
*/

#include <stdio.h>
#include <stdlib.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime_api.h>

#include "common.cuh"
#include "filters.cuh"
#include "coeff.cuh"

hipfftComplex* filter48k;
hipfftComplex* filter96k;
hipfftComplex* filter192k;
hipfftComplex* filter384k;

hipfftComplex* deviceFilter48k;
hipfftComplex* deviceFilter96k;
hipfftComplex* deviceFilter192k;
hipfftComplex* deviceFilter384k;

//static hipfftHandle      planR2C;

__global__ void	gpu_make_analytic(hipfftComplex*);

void generateFilter(double samplingrate,double Fb,hipfftComplex* deviceFilter,hipfftComplex* filter);

void loadFilters(int pcie) {
    
fprintf(stderr,"filter size=%ld\n",(COMPLEX_SIGNAL_SIZE+1)*sizeof(hipfftComplex));

    hipError_t result = hipHostAlloc(&filter48k, (COMPLEX_SIGNAL_SIZE+1)*sizeof(hipfftComplex), hipHostMallocMapped);
    if (result != hipSuccess) {
       fprintf(stderr, "Error filter 48k hipHostAlloc %d\n", result);
       exit(EXIT_FAILURE);
    }
    result = hipHostGetDevicePointer(&deviceFilter48k, filter48k, 0);
    if (result != hipSuccess) {
       fprintf(stderr, "Error filter 48k hipHostGetDevicePointer %d\n", result);
       exit(EXIT_FAILURE);
    }
    generateFilter(samplingrate,48000.0,deviceFilter48k,filter48k);

    result = hipHostAlloc(&filter96k, (COMPLEX_SIGNAL_SIZE+1)*sizeof(hipfftComplex), hipHostMallocMapped);
    if (result != hipSuccess) {
       fprintf(stderr, "Error filter 96k hipHostAlloc %d\n", result);
       exit(EXIT_FAILURE);
    }
    result = hipHostGetDevicePointer(&deviceFilter96k, filter96k, 0);
    if (result != hipSuccess) {
       fprintf(stderr, "Error filter 96k hipHostGetDevicePointer %d\n", result);
       exit(EXIT_FAILURE);
    }
    generateFilter(samplingrate,96000.0,deviceFilter96k,filter96k);

    result = hipHostAlloc(&filter192k, (COMPLEX_SIGNAL_SIZE+1)*sizeof(hipfftComplex), hipHostMallocMapped);
    if (result != hipSuccess) {
       fprintf(stderr, "Error filter 192k hipHostAlloc %d\n", result);
       exit(EXIT_FAILURE);
    }
    result = hipHostGetDevicePointer(&deviceFilter192k, filter192k, 0);
    if (result != hipSuccess) {
       fprintf(stderr, "Error filter 192k hipHostGetDevicePointer %d\n", result);
       exit(EXIT_FAILURE);
    }
    generateFilter(samplingrate,192000.0,deviceFilter192k,filter192k);

    result = hipHostAlloc(&filter384k, (COMPLEX_SIGNAL_SIZE+1)*sizeof(hipfftComplex), hipHostMallocMapped);
    if (result != hipSuccess) {
       fprintf(stderr, "Error filter 384k hipHostAlloc %d\n", result);
       exit(EXIT_FAILURE);
    }
    result = hipHostGetDevicePointer(&deviceFilter384k, filter384k, 0);
    if (result != hipSuccess) {
       fprintf(stderr, "Error filter 384k hipHostGetDevicePointer %d\n", result);
       exit(EXIT_FAILURE);
    }
    generateFilter(samplingrate,384000.0,deviceFilter384k,filter384k);
}

void generateFilter(double samplingrate,double Fb,hipfftComplex* deviceFilter,hipfftComplex* filter) {

    hipfftReal* coeff;
    hipfftReal* deviceCoeff;
    hipfftHandle planR2C;

fprintf(stderr,"coeffs size=%ld\n",DFT_BLOCK_SIZE*sizeof(hipfftReal));
    hipError_t result = hipHostAlloc(&coeff, DFT_BLOCK_SIZE*sizeof(hipfftReal), hipHostMallocMapped);
    if (result != hipSuccess) {
       fprintf(stderr, "Error coeff hipHostAlloc %d\n", result);
       exit(EXIT_FAILURE);
    }

    result = hipHostGetDevicePointer(&deviceCoeff, coeff, 0);
    if (result != hipSuccess) {
       fprintf(stderr, "Error coeff hipHostGetDevicePointer %d\n", result);
       exit(EXIT_FAILURE);
    }

    double *H=calcFilter(samplingrate,0.0,Fb,P_SIZE,25.0);
    for(int i=0;i<P_SIZE;i++) {
        coeff[i]=(hipfftReal)H[i];
    }
    free(H);

    for(int i=P_SIZE; i<DFT_BLOCK_SIZE;i++) {
        coeff[i]=0.0;
    }

#ifdef DUMPDATA
for(int i=0;i<1024;i++) {
    fprintf(stderr,"coeff %d=%.24f\n", i, coeff[i]);
}
#endif

    hipfftResult error = hipfftPlan1d(&planR2C, DFT_BLOCK_SIZE, HIPFFT_R2C, 1);
    if(error!=HIPFFT_SUCCESS) {
       fprintf(stderr,"Error creating hipfftPlan1d for FIR: %s\n", _cudaGetErrorEnum(error));
       exit(EXIT_FAILURE);
    }

    error = hipfftExecR2C(planR2C, deviceCoeff, deviceFilter);
    if (error != HIPFFT_SUCCESS) {
       fprintf(stderr, "Error hipfftExecR2C (planR2C) %d\n", result);
       exit(EXIT_FAILURE);
    }

    gpu_make_analytic<<<COMPLEX_SIGNAL_SIZE/4096, 1024>>>(deviceFilter);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if(err != hipSuccess)
    {
      // print the CUDA error message and exit
      fprintf(stderr,"gpu_make_analytic error: %s\n", hipGetErrorString(err));
      exit(-1);
    }


#ifdef DUMPDATA
for(int i=0;i<1024;i++) {
    fprintf(stderr,"analytic %d=%.24f:%.24f\n", i, filter[i].x, filter[i].y);
}
#endif

    error = hipfftDestroy(planR2C);
    if (error != HIPFFT_SUCCESS) {
       fprintf(stderr, "Error hipfftDestroy (planR2C) %d\n", result);
       exit(EXIT_FAILURE);
    }

    result = hipHostFree(coeff);
    if (result != hipSuccess) {
       fprintf(stderr, "Error coeff hipHostFree %d\n", result);
       exit(EXIT_FAILURE);
    }
}

hipfftComplex* getFilter(int rate) {
    hipfftComplex* filter;
    filter=(hipfftComplex*)0;
    switch(rate) {
        case 0:
           filter=filter48k;
           break;
        case 1:
           filter=filter96k;
           break;
        case 2:
           filter=filter192k;
           break;
        case 3:
           filter=filter384k;
           break;
        default:
           fprintf(stderr,"getFilter: invalid rate %d/n", rate);
           exit(EXIT_FAILURE);
           break;
    }
    return filter;
}

hipfftComplex* getDeviceFilter(int rate) {
    hipfftComplex* filter;
    filter=(hipfftComplex*)0;
    switch(rate) {
        case 0:
           filter=deviceFilter48k;
           break;
        case 1:
           filter=deviceFilter96k;
           break;
        case 2:
           filter=deviceFilter192k;
           break;
        case 3:
           filter=deviceFilter384k;
           break;
        default:
           fprintf(stderr,"getDeviceFilter: invalid rate %d/n", rate);
           exit(EXIT_FAILURE);
           break;
    }
    return filter;
}

