#include "hip/hip_runtime.h"
/**
* @file receiver.cu
* @brief Implement a receiver
* @author John Melton, G0ORX/N6LYT
*/


/* Copyright (C)
* 2015 - John Melton, G0ORX/N6LYT
*
* Based on code by Steven Passe AD0ES and Vasiliy Gokoyev K3IT
*
* This program is free software; you can redistribute it and/or
* modify it under the terms of the GNU General Public License
* as published by the Free Software Foundation; either version 2
* of the License, or (at your option) any later version.
*
* This program is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
* GNU General Public License for more details.
*
* You should have received a copy of the GNU General Public License
* along with this program; if not, write to the Free Software
* Foundation, Inc., 59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.
*
*/

#include <errno.h>
#include <pthread.h>
#include <sched.h>
#include <semaphore.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <sys/ioctl.h>
#include <sys/socket.h>
#include <arpa/inet.h>
#include <net/if.h>
#include <netinet/if_ether.h>
#include <netpacket/packet.h>
#include <net/if_packet.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime_api.h>

#include "common.cuh"
#include "receiver.cuh"
#include "dfc.cuh"
#include "inputbuffer.cuh"
#include "rawbuffer.cuh"
#include "filters.cuh"
#include "hermes.cuh"
#include "time.cuh"

#define SCALE_FACTOR  8388607.0

float scale_factor=0.25F;

RECEIVER receiver[MAX_RECEIVER];
void* receiverThread(void* arg);

void initReceiver(int rx) {
    int result;
    hipError_t error;
    RECEIVER* r;

    fprintf(stderr,"initReceiver %d: scale_factor=%f\n",rx,scale_factor);
    r=&receiver[rx];

    r->id=rx;

    error = hipHostAlloc(&r->receiverdata, COMPLEX_SIGNAL_SIZE*sizeof(hipfftComplex), hipHostMallocMapped);
    if (error != hipSuccess) {
       fprintf(stderr, "initReceiver: Error hipHostAlloc for receiver data %d\n", error);
       exit(EXIT_FAILURE);
    }

    error = hipHostGetDevicePointer(&(r->deviceReceiverdata), r->receiverdata, 0);
    if (error != hipSuccess) {
       fprintf(stderr, "initReceiver: Error receiverdata hipHostGetDevicePointer %d\n", error);
       exit(EXIT_FAILURE);
    }


    //fprintf(stderr,"slice size=%d\n",(COMPLEX_SIGNAL_SIZE/D_SIZE_384K)*sizeof(hipfftComplex));
    error = hipHostAlloc(&r->slice, (COMPLEX_SIGNAL_SIZE/D_SIZE_384K)*sizeof(hipfftComplex), hipHostMallocMapped);
    if (error != hipSuccess) {
        fprintf(stderr, "processReceiverData: Error hipHostAlloc for slice data %d\n", error);
        exit(EXIT_FAILURE);
    }

    error = hipHostGetDevicePointer(&r->deviceSlice, r->slice, 0);
    if (error != hipSuccess) {
       fprintf(stderr, "processReceiveData: Error slice data hipHostGetDevicePointer %d\n", error);
       exit(EXIT_FAILURE);
    }


    //fprintf(stderr,"RX_TD_MAXSIZE=%d\n",RX_TD_MAXSIZE);
    //fprintf(stderr,"decimate size=%d\n",(int)(RX_TD_MAXSIZE*sizeof(hipfftComplex)));
    error = hipHostAlloc(&r->decimate, RX_TD_MAXSIZE*sizeof(hipfftComplex), hipHostMallocMapped);
    if (error != hipSuccess) {
       fprintf(stderr, "processReceiverData: Error hipHostAlloc for decimate %d\n", error);
       exit(EXIT_FAILURE);
    }

    error = hipHostGetDevicePointer(&r->deviceDecimate, r->decimate, 0);
    if (error != hipSuccess) {
       fprintf(stderr, "processReceiveData: Error decimate hipHostGetDevicePointer %d\n", error);
       exit(EXIT_FAILURE);
    }

    //fprintf(stderr,"tdoutput size=%d\n",(int)(RX_TD_MAXSIZE*sizeof(hipfftComplex)));
    error = hipHostAlloc(&r->tdOutput, RX_TD_MAXSIZE*sizeof(hipfftComplex), hipHostMallocMapped);
    if (error != hipSuccess) {
        fprintf(stderr, "initHermes: Error hipHostAlloc for td output data %d\n", error);
        exit(EXIT_FAILURE);
    }

    error = hipHostGetDevicePointer(&r->deviceTdOutput, r->tdOutput, 0);
    if (error != hipSuccess) {
        fprintf(stderr, "initHermes: Error td output hipHostGetDevicePointer %d\n", error);
        exit(EXIT_FAILURE);
    }

    //fprintf(stderr,"output size=%d\n",(int)(RX_TD_MAXSIZE*sizeof(char)*6));
    error = hipHostAlloc(&r->output, RX_TD_MAXSIZE*sizeof(char)*6, hipHostMallocMapped);
    if (error != hipSuccess) {
        fprintf(stderr, "initHermes: Error hipHostAlloc for output data %d\n", error);
        exit(EXIT_FAILURE);
    }

    error = hipHostGetDevicePointer(&r->deviceOutput, r->output, 0);
    if (error != hipSuccess) {
        fprintf(stderr, "initHermes: Error output hipHostGetDevicePointer %d\n", error);
        exit(EXIT_FAILURE);
    }

    result=sem_init(&r->inputReady, 0, 0);
    if(result!=0) {
        fprintf(stderr,"initReceiver %d: sem_init failed for inputReady%d\n", rx, result);
        exit(EXIT_FAILURE);
    }

    result=sem_init(&r->outputReady, 0, 0);
    if(result!=0) {
        fprintf(stderr,"initReceiver %d: sem_init failed for outputReady%d\n", rx, result);
        exit(EXIT_FAILURE);
    }


    if((result=pthread_create(&r->receiverThreadId, NULL, receiverThread, r)) < 0) {
        fprintf(stderr, "receiverThread create failed %d\n",result);
        exit(EXIT_FAILURE);
    }

}

__global__ void
gpu_mix_and_convolve(const hipfftComplex* d_fft, const hipfftComplex* d_fir_fft,
                     hipfftComplex* d_receiver, const int nrot,
                     const float scale, int d_size)
{
    const size_t numThreads = blockDim.x * gridDim.x;
    const size_t tid = blockIdx.x * blockDim.x + threadIdx.x;

    size_t new_index;

    for (int i = tid; i < COMPLEX_SIGNAL_SIZE; i += numThreads) {
        new_index = (i >= nrot) ? i - nrot : COMPLEX_SIGNAL_SIZE - nrot + i;

        // Skip computing unneeded bins.
        if (new_index > COMPLEX_SIGNAL_SIZE / d_size)
            continue;

        d_receiver[new_index] = ComplexScale(ComplexMul(d_fft[i], d_fir_fft[new_index]), scale);
    }
}

__global__ void
gpu_decimate(const hipfftComplex* deviceReceiver, hipfftComplex* deviceSlice, int d_size, int outrot) {
    const int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    size_t new_index;

    new_index = (threadId >= outrot) ? threadId - outrot : COMPLEX_SIGNAL_SIZE - outrot + threadId;
    deviceSlice[threadId] = deviceReceiver[new_index];
}

__global__ void
gpu_ifft_postprocess(const hipfftComplex* d_slice, hipfftComplex* d_rx_td,
                     char* d_rx_td_24bit, int decimate, int d_size, int rx_td_size,float scale_factor
                    )
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= rx_td_size)
        return;

    int idx = tid * decimate + (P_SIZE-1)/2/d_size;

    d_rx_td[tid]=d_slice[idx];

    // Note: I & Q must be swapped.
    long tempQ = (long)((double)d_rx_td[tid].x * scale_factor); //* SCALE_FACTOR);
    long tempI = (long)((double)d_rx_td[tid].y * scale_factor); //* SCALE_FACTOR);


    // Load samples in big endian format.
    int baseindex = tid * 6;            // start of the 24 bit sample
    d_rx_td_24bit[baseindex++] = (char)((tempI >> 16) & 0xff);
    d_rx_td_24bit[baseindex++] = (char)((tempI >> 8) & 0xff);
    d_rx_td_24bit[baseindex++] = (char)((tempI >> 0) & 0xff);
    d_rx_td_24bit[baseindex++] = (char)((tempQ >> 16) & 0xff);
    d_rx_td_24bit[baseindex++] = (char)((tempQ >> 8) & 0xff);
    d_rx_td_24bit[baseindex++] = (char)((tempQ >> 0) & 0xff);

}

void* receiverThread(void* arg) {
    int result;
    hipError_t error;
    RECEIVER* r=(RECEIVER*)arg;

#ifdef TIMING
    long long starttime;
    long long endtime;
#endif

    fprintf(stderr,"receiverThread %d: running on cpu %d\n", r->id, sched_getcpu());

    while(1) {
        result=sem_wait(&r->inputReady);
        if(result!=0) {
            fprintf(stderr, "receiverThread: sem_wait failed for inputReady: %d\n", result);
            exit(EXIT_FAILURE);
        }

//fprintf(stderr,"gpu_mix_and_convolve<<<%d,%d>>> rx=%d rotate=%d, scale=%f, d_size=%d\n", COMPLEX_SIGNAL_SIZE/8192,1024,r->id,r->rotate,r->scale,r->d_size);

        gpu_mix_and_convolve<<<COMPLEX_SIGNAL_SIZE/8192, 1024>>>
            (deviceFrequencysamples, r->deviceFilter, r->deviceReceiverdata, r->rotate, r->scale, r->d_size);
/*
    hipDeviceSynchronize();
    error = hipGetLastError();
    if(error != hipSuccess) {
      // print the CUDA error message and exit
      fprintf(stderr,"gpu_mix_and_convolve CUDA error: %s\n", hipGetErrorString(error));
      exit(-1);
    }
*/

        //gpu_decimate<<<COMPLEX_SIGNAL_SIZE/1024/r->d_size, 1024>>>
        gpu_decimate<<<COMPLEX_SIGNAL_SIZE/8192, 1024>>>
            (r->deviceReceiverdata, r->deviceSlice, r->d_size, r->outrot );
/*
    hipDeviceSynchronize();
    error = hipGetLastError();
    if(error != hipSuccess) {
      // print the CUDA error message and exit
      fprintf(stderr,"gpu_decimate CUDA error: %s\n", hipGetErrorString(error));
      exit(-1);
    }
*/

        // inverse FFT
        hipfftResult err=hipfftExecC2C(r->planC2C, r->deviceSlice, r->deviceSlice, HIPFFT_BACKWARD);
        if(err!=HIPFFT_SUCCESS) {
           fprintf(stderr,"Error executing planC2C for input buffer: %s\n", _cudaGetErrorEnum(err));
           exit(EXIT_FAILURE);
        }

        // convert to 24 bit samples
        gpu_ifft_postprocess<<<r->rx_td_size/1024 + 1, 1024>>>
            (r->deviceSlice, r->deviceTdOutput, r->deviceOutput, r->ifft_decimate_factor, r->d_size, r->rx_td_size, scale_factor);

        // need to sync as last stage
        hipDeviceSynchronize();
        error = hipGetLastError();
        if(error != hipSuccess) {
          // print the CUDA error message and exit
          fprintf(stderr,"gpu_ifft_postprocess CUDA error: %s\n", hipGetErrorString(error));
          exit(-1);
        }

        result=sem_post(&r->outputReady);
        if(result!=0) {
            fprintf(stderr, "receiverThread: sem_post failed for outputReady: %d\n", result);
            exit(EXIT_FAILURE);
        }

    }
}
