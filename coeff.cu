
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

static double Ino(double x) {
  /*
   * This function calculates the zeroth order Bessel function
   */
  double d = 0.0, ds = 1.0, s = 1.0;
  do
  {
    d += 2.0;
    ds *= x*x/(d*d);
    s += ds;
  }
  while (ds > s*1e-6);
  return s;
}

double *calcFilter(double Fs, double Fa, double Fb, int M, double Att) {

  int Np = (M-1)/2;
  double A[Np+1];
  double Alpha;
  int j;
  double pi = 3.1415926535897932;
  double Inoalpha;
  double *H;

  H=(double*)malloc(M*sizeof(double));
    // Calculate the impulse response of the ideal filter
    A[0] = 2.0*(Fb-Fa)/Fs;
    for(j=1; j<=Np; j++)
    {
      A[j] = (sin(2.0*(double)j*pi*Fb/Fs)-sin(2.0*(double)j*pi*Fa/Fs))/((double)j*pi);
    }
    // Calculate the desired shape factor for the Kaiser-Bessel window
    if (Att<21.0)
    {
      Alpha = 0.0;
    }
    else if (Att>50.0)
    {
      Alpha = 0.1102*(Att-8.7);
    }
    else
    {
      Alpha = 0.5842*pow((Att-21.0), 0.4)+0.07886*(Att-21.0);
    }
    // Window the ideal response with the Kaiser-Bessel window
    Inoalpha = Ino(Alpha);
    for (j=0; j<=Np; j++)
    {
      H[Np+j] = A[j]*Ino(Alpha*sqrt(1.0-((double)j*(double)j/((double)Np*(double)Np))))/Inoalpha;
    }
    for (j=0; j<Np; j++)
    {
      H[j] = H[M-1-j];
    }

    return H;
}

